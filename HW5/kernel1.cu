#include "hip/hip_runtime.h"
#ifdef _WIN32
#include <hip/hip_runtime.h>
#include <>
#else
#include <hip/hip_runtime.h>
#endif

#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#include "kernel.h"

using namespace std;

#if USE_KERNEL == 1

// 1600 * 1200
#define GRID_X 100
#define GRID_Y 75
#define BLOCK_X 16
#define BLOCK_Y 16

static int cudaInited;

void cudaInit()
{
    hipError_t cudaStatus;

    if (cudaInited)
        return;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        cerr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?" << endl;
        exit(EXIT_FAILURE);
    }

    cudaInited = 1;
}

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;

    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(int *output, float x0, float y0, float dx, float dy, int maxIterations)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float x = x0 + i * dx;
    float y = y0 + j * dy;
    int index = j * gridDim.x * blockDim.x + i;
    output[index] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    hipError_t cudaStatus;
    int *cudaResult, *result;
    float dx, dy;

    cudaInit();

    cudaStatus = hipMalloc((void **)&cudaResult, sizeof(int) * resX * resY);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        exit(EXIT_FAILURE);
    }

    // HW required
    result = (int *)malloc(sizeof(int) * resX * resY);
    
    dx = (upperX - lowerX) / resX;
    dy = (upperY - lowerY) / resY;

    dim3 dimGrid(GRID_X, GRID_Y);
    dim3 dimBlock(BLOCK_X, BLOCK_Y);

    mandelKernel<<<dimGrid, dimBlock>>>(cudaResult, lowerX, lowerY, dx, dy, maxIterations);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching addKernel!" << endl;
        exit(EXIT_FAILURE);
    }

    cudaStatus = hipMemcpy(result, cudaResult, sizeof(int) * resX * resY, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        exit(EXIT_FAILURE);
    }

    hipFree(cudaResult);

    // Copy result to output
    memcpy(img, result, sizeof(int) * resX * resY);

    free(result);
}

#endif