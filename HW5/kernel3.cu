#include "hip/hip_runtime.h"
#ifdef _WIN32
#include <hip/hip_runtime.h>
#include <>
#else
#include <hip/hip_runtime.h>
#endif

#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#include "kernel.h"

using namespace std;

#if USE_KERNEL == 3

// Group size: 4
// 1600 * 1200 -> 1600 * 300
#define GRID_X 100
#define GRID_Y 5
#define BLOCK_X 16
#define BLOCK_Y 60

static int cudaInited;

void cudaInit()
{
    hipError_t cudaStatus;

    if (cudaInited)
        return;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        cerr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?" << endl;
        exit(EXIT_FAILURE);
    }

    cudaInited = 1;
}

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;

    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(int *output, float x0, float y0, float dx, float dy, int maxIterations)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = (blockIdx.y * blockDim.y + threadIdx.y) * 4;
    float x = x0 + i * dx;

    #pragma unroll 4
    for (int loop = 0; loop < 4; ++loop) {
        float y = y0 + (j + loop) * dy;
        int index = (j + loop) * gridDim.x * blockDim.x + i;
        output[index] = mandel(x, y, maxIterations);
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    hipError_t cudaStatus;
    int *cudaResult, *result;
    size_t pitch;
    float dx, dy;

    cudaInit();

    cudaStatus = hipMallocPitch((void **)&cudaResult, &pitch, sizeof(int) * resX, resY);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMallocPitch failed!" << endl;
        exit(EXIT_FAILURE);
    }

    // HW required
    cudaStatus = hipHostAlloc((void **)&result, sizeof(int) * resX * resY, hipHostMallocDefault);
    if (cudaStatus != hipSuccess) {
        cerr << "hipHostAlloc failed!" << endl;
        exit(EXIT_FAILURE);
    }

    dx = (upperX - lowerX) / resX;
    dy = (upperY - lowerY) / resY;

    dim3 dimGrid(GRID_X, GRID_Y);
    dim3 dimBlock(BLOCK_X, BLOCK_Y);

    mandelKernel<<<dimGrid, dimBlock>>>(cudaResult, lowerX, lowerY, dx, dy, maxIterations);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching addKernel!" << endl;
        exit(EXIT_FAILURE);
    }

    cudaStatus = hipMemcpy(result, cudaResult, sizeof(int) * resX * resY, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        exit(EXIT_FAILURE);
    }

    hipFree(cudaResult);

    // Copy result to output
    memcpy(img, result, sizeof(int) * resX * resY);

    hipHostFree(result);
}

#endif